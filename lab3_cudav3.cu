#include "hip/hip_runtime.h"
#include "lab3_cuda.h"
#include <iostream>
#include <cmath>
#include <malloc.h>
#include <fstream>
#include <bits/stdc++.h>
#include <hip/hip_runtime.h>


#define pb push_back

using namespace std;

#define TOLERANCE 0.001
#define JACOBI_UPDATE_TOLERANCE 0.001
#define FILENAME1 "testcase_1000_300"
#define FILENAME2 "iris_stndardized"
#define samples 150
#define features 4
#define BLOCK_SIZE 16

double **S; //Symmetric matrix (input)
double  *e; //eigenvalues
double **E; //eigenvectors
int  *ind;
bool *changed;
int  state;
int  N;

void read_file(char* filename, int num_samples, int num_features, double** A) {
    ifstream ifile;
    ifile.open(filename, ios::in);

    double tmp;
    for (int i=0; i<num_samples; i++) {
        for (int j=0; j<num_features; j++){
            ifile >> tmp;
            A[i][j] = tmp;
        }
    }

    ifile.close();
}

double* mat_transpose(double* A, int Am, int An) {
    double *B;
    B = (double*)malloc(__SIZEOF_DOUBLE__*An*Am);
    // B = (double**)malloc(__SIZEOF_POINTER__*An);
    // for (int i=0; i<An; i++)
    //     B[i] = (double*)malloc(__SIZEOF_DOUBLE__*Am);

    for (int i=0; i<Am; i++){
        for (int j=0; j<An; j++){
            B[j*Am + i] = A[i*An + j];
        }
    }

    return B;
}

double** mat_mul(double** A, int Am, int An, 
                 double** B, int Bm, int Bn){
    double **C;
    C = (double**)malloc(__SIZEOF_POINTER__*Am);
    for (int i=0; i<Am; i++)
        C[i] = (double*)malloc(__SIZEOF_DOUBLE__*Bn);

    for (int i=0; i<Am; i++){
        for (int j=0; j<Bn; j++){
            C[i][j] = 0;
            for (int k=0; k<An; k++){
                C[i][j] += A[i][k] * B[k][j];
            }
        }
    }

    return C;
}


double* new_mat_mul(double* A, int Am, int An, 
                 double* B, int Bm, int Bn){
    double *C;
    C = (double*)malloc(__SIZEOF_DOUBLE__*Am*Bn);

    // C = (double**)malloc(__SIZEOF_POINTER__*Am);
    // for (int i=0; i<Am; i++)
    //     C[i] = (double*)malloc(__SIZEOF_DOUBLE__*Bn);

    for (int i=0; i<Am; i++){
        for (int j=0; j<Bn; j++){
            C[i*Bn + j] = 0;
            for (int k=0; k<An; k++){
                C[i*Bn + j] += A[i*An + k] * B[k*Bn + j];
            }
        }
    }

    return C;
}

// dim3 dimGrid((k + BLOCK_SIZE - 1) / BLOCK_SIZE, (m + BLOCK_SIZE - 1) / BLOCK_SIZE);
// dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

__global__ void gpu_matmul(double *a,double *b, double *c, int m, int n, int k)
{ 
    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    double sum = 0;
    if( col < k && row < m) 
    {
        for(int i = 0; i < n; i++) 
        {
            sum += a[row * n + i] * b[i * k + col];
        }
        c[row * k + col] = sum;
    }
} 

int maxind(int k) {
    int m = k+1;

    for (int i = k+2; i < N; i++){
        if (fabs(S[k][i]) > fabs(S[k][m])){
            m = i;
        }
    }

    return m;
}

void update(int k, double t) {
    double ek_prev = e[k];
    e[k] = ek_prev + t;

    if (e[k] < 0) e[k] = 0;

    if (changed[k] && (ek_prev - e[k]) < JACOBI_UPDATE_TOLERANCE) {
        changed[k] = false;
        state = state - 1;
    }
    else if ((! changed[k]) && (ek_prev - e[k]) > JACOBI_UPDATE_TOLERANCE) {
        changed[k] = true;
        state = state + 1;
    }
}

void rotate(int k, int l, int i, int j, double c, double s,
            bool eigenvectors){
    // double** mat1;
    // double** mat2;
    // double** mat3;

    // double mat1_00;
    // double mat1_01;
    // double mat1_10;
    // double mat1_11;


    double mat2_00;
    double mat2_10;

    // double mat3_00;
    // double mat3_10;

    // mat1 = (double**)malloc(__SIZEOF_POINTER__*2);
    // mat1[0] = (double*)malloc(__SIZEOF_DOUBLE__*2);
    // mat1[1] = (double*)malloc(__SIZEOF_DOUBLE__*2);
    // mat1[0][0] = c; mat1[0][1] = -s;
    // mat1[1][0] = s; mat1[1][1] = c;

    // mat1_00 = c; mat1_01 = -s;
    // mat1_10 = s; mat1_11 = c;

    // mat2 = (double**)malloc(__SIZEOF_POINTER__*2);
    // mat2[0] = (double*)malloc(__SIZEOF_DOUBLE__*1);
    // mat2[1] = (double*)malloc(__SIZEOF_DOUBLE__*1);
    // if (eigenvectors){
    //     mat2[0][0] = E[i][k];
    //     mat2[1][0] = E[i][l];
    // }
    // else {
    //     mat2[0][0] = S[k][l];
    //     mat2[1][0] = S[i][j];
    // }

    if (eigenvectors){
        mat2_00 = E[i][k];
        mat2_10 = E[i][l];
    }
    else {
        mat2_00 = S[k][l];
        mat2_10 = S[i][j];
    }

    // mat3_00 = (c*mat2_00) - (s*mat2_10);
    // mat3_10 = (s*mat2_00) + (c*mat2_10);

    // mat3 = mat_mul(mat1, 2, 2, mat2, 2, 1);

    // if (eigenvectors){
    //     E[i][k] = mat3[0][0];
    //     E[i][l] = mat3[1][0];
    // }
    // else{
    //     S[k][l] = mat3[0][0];
    //     S[i][j] = mat3[1][0];
    // }

    if (eigenvectors){
        E[i][k] = (c*mat2_00) - (s*mat2_10);;
        E[i][l] = (s*mat2_00) + (c*mat2_10);;
    }
    else{
        S[k][l] = (c*mat2_00) - (s*mat2_10);;
        S[i][j] = (s*mat2_00) + (c*mat2_10);;
    }


    // free(mat1[0]);
    // free(mat1[1]);
    // free(mat1);
    // free(mat2[0]);
    // free(mat2[1]);
    // free(mat2);
    // free(mat3[0]);
    // free(mat3[1]);
    // free(mat3);

}

void print_matrix(double** A, int Am, int An) {
    cout << "[";
    for (int i=0; i<Am; i++){
        if (i>0)
            cout<<" ";
        cout<<"[";
        for (int j=0; j<An-1; j++){
            cout << A[i][j] << ", ";
        }
        if (i < Am-1)
            cout << A[i][An-1] << "]" << endl;
    }
    cout << A[Am-1][An-1] << "]]" << endl;
}

void print_vector(double* A, int An) {
    cout << "[";
    for(int i=0; i<An-1; i++)
        cout << A[i] << ",";
    cout << A[An-1] << "]" << endl;
}

void init_jacobi() {
    E = (double**)malloc(__SIZEOF_POINTER__*N);
    for (int i=0; i<N; i++){
        E[i] = (double*)malloc(__SIZEOF_DOUBLE__*N);
        for (int j=0; j<N; j++){
            E[i][j] = 0;
        }
        E[i][i] = 1;
    }

    state = N;

    e = (double*)malloc(__SIZEOF_DOUBLE__*N);
    ind = (int*)malloc(__SIZEOF_INT__*N);
    changed = (bool*)malloc(sizeof(bool)*N);

    for (int k=0; k<N; k++){
        ind[k]     = maxind(k);
        e[k]       = S[k][k];
        changed[k] = true;
    }
}

void Jacobi(double **input_matrix, int n, 
            double **eigenvalues, double ***eigenvectors) {
    N = n;
    S = input_matrix;

    init_jacobi();

    while(state != 0){
        int m = 0;

        for (int k=1; k<N-1; k++){
            if (fabs(S[k][ind[k]]) > fabs(S[m][ind[m]])){
                m = k;
            }
        }

        int k = m;
        int l = ind[m];
        double p = S[k][l];
        double y = (e[l] - e[k]) / 2.0;
        double d = fabs(y) + sqrt(p*p + y*y);
        double r = sqrt(p*p + d*d);
        double c = d / r;
        double s = p / r;
        double t = (p*p) / d;

        if (y < 0.0) { s = -s; t = -t; }

        S[k][l] = 0.0;
        update(k, -t);
        update(l, t);

        for (int i=0; i<k; i++)  { rotate(i, k, i, l, c, s, false); }
        for (int i=k+1; i<l; i++){ rotate(k, i, i, l, c, s, false); }
        for (int i=l+1; i<N; i++)  { rotate(k, i, l, i, c, s, false); }

        for (int i=0; i<N; i++){
            rotate(k, l, i, i, c, s, true);
        }

        ind[k] = maxind(k);
        ind[l] = maxind(l);
    }

    *eigenvalues = e;
    *eigenvectors = E;
}

// int main(){
//     double **D, **D_T;
//     double **prod, *eigenvalues, **eigenvectors;

//     D = (double**)malloc(sizeof(double*)*samples);
//     for (int i=0; i<samples; i++)
//         D[i] = (double*)malloc(sizeof(double)*features);

//     read_file((char*)FILENAME1, samples, features, D);

//     D_T = mat_transpose(D, samples, features);

//     prod = mat_mul(D_T, features, samples, D, samples, features);
//     Jacobi(prod, features, &eigenvalues, &eigenvectors);

//     cout << "\neigenvalues:" << endl;
//     print_vector(eigenvalues, features);

//     cout << "\neigenvectors:" << endl;
//     print_matrix(eigenvectors, features, features);

//     return 0;
// }

// /*
// 	*****************************************************
// 		TODO -- You must implement this function
// 	*****************************************************
// */
void SVD_and_PCA (int M, 
        int N, 
        double* D, 
        double** U, 
        double** SIGMA, 
        double** V_T, 
        double** D_HAT, 
        int *K,
        int retention) {
    // write your code here

	double *d;
	double *d_t;
    double **product, *eigenvalues, **eigenvectors;
    // double **v;


    d = (double*)malloc(sizeof(double*)*M*N);

    // 
    // for (int i=0; i<M; i++)
    //     d[i] = (double*)malloc(sizeof(double)*N);


    for(int i=0;i<M;i++){
    	for(int j=0;j<N;j++) d[i*N+j] = D[i*N+j];
    }

    d_t = mat_transpose(d, M, N);

    // for(int i=0;i<N;i++){
    //   for(int j=0;j<M;j++){
    //       printf("%f ", (d_t)[i*M+j]);
    //   }
    //   printf("\n");
    // }


    // product = new_mat_mul(d_t, N, M, d, M, N);

    // for(int i=0;i<N;i++){
    //   for(int j=0;j<N;j++){
    //       printf("%f ", (product)[i*N+j]);
    //   }
    //   printf("\n");
    // }

///////////////////////////////////////////////////////////////
    double *product1;
    product1 = (double*)malloc(sizeof(double)*N*N);
    
    double *gpu_a, *gpu_b, *gpu_c;
    hipMalloc((void **) &gpu_a, sizeof(double)*N*M);
    hipMalloc((void **) &gpu_b, sizeof(double)*M*N);
    hipMalloc((void **) &gpu_c, sizeof(double)*N*N);

    // copying matrix d_t and d from host to device memory
    hipMemcpy(gpu_a, d_t, sizeof(double)*N*M, hipMemcpyHostToDevice);
    hipMemcpy(gpu_b, d, sizeof(double)*M*N, hipMemcpyHostToDevice);
    
    unsigned int g_rows = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int g_cols = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(g_cols, g_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    
    gpu_matmul<<<dimGrid, dimBlock>>>(gpu_a, gpu_b, gpu_c, N, M, N);
    hipMemcpy(product1, gpu_c, sizeof(double)*N*N, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    // for(int i=0;i<N;i++){
    //   for(int j=0;j<N;j++){
    //       printf("%f ", (product1)[i*N+j]);
    //   }
    //   printf("\n");
    // }

///////////////////////////////////////////////////////////////

    // size N*N
    product = (double**)malloc(sizeof(double*)*N);
    for (int i=0; i<N; i++)
        product[i] = (double*)malloc(sizeof(double)*N);

    for(int i=0; i<N; i++){
        for(int j=0; j<N; j++) product[i][j]=product1[i*N + j];
    }

    // for(int i=0;i<N;i++){
    //   for(int j=0;j<N;j++){
    //       printf("%f ", (product)[i][j]);
    //   }
    //   printf("\n");
    // }


    float computation_time1;
    hipEvent_t start1, stop1;
    hipEventCreate(&start1);
    hipEventCreate(&stop1);

    hipEventRecord(start1);

    Jacobi(product, N, &eigenvalues, &eigenvectors);

    hipEventRecord(stop1);
    hipEventSynchronize(stop1);
    hipEventElapsedTime(&computation_time1, start1, stop1);
    printf("Time taken for Jacobi: %f\n", computation_time1);   


    // for(int i=0;i<N;i++) printf("%f\n", eigenvalues[i]);

    vector<double> eigenvals;
    for(int i=0; i<N; i++) eigenvals.pb(eigenvalues[i]);

	vector<pair<double, int> > eigenv_index;
	
	for(int i=0; i<eigenvals.size(); i++){
		eigenv_index.pb(make_pair(eigenvalues[i],i));
	}

	sort(eigenv_index.begin(), eigenv_index.end());

	int e = eigenv_index.size()-1;
	
 	for(int i=0;i<N;i++){
		(*SIGMA)[i] = sqrt(eigenv_index[e].first);
		e--;
	}


    // for(int i=0;i<N;i++) printf("%f\n", (*SIGMA)[i]);

    double *u = (double*)malloc(sizeof(double)*N*N);
	// double **u = (double**)malloc(sizeof(double*)*N);
 //    for (int i=0; i<N; i++)
 //        u[i] = (double*)malloc(sizeof(double)*N);


	e = eigenv_index.size()-1;	
	for(int j=0;j<N;j++){
		int index = eigenv_index[e].second;
		for(int i=0;i<N;i++){
			u[i*N + j] = eigenvectors[i][index];
		}
		e--;
	}

	for(int j=0;j<N;j++){
		for(int i=0;i<N;i++){
			(*U)[i*N+j] = u[i*N + j];
		}
	}


	// for(int j=0;j<N;j++){
	// 	for(int i=0;i<N;i++){
	// 		printf("%f ", (*U)[i*N+j]);
	// 	}
	// 	printf("\n");
	// }


	// size N*M
    double *sigma_invT = (double*)malloc(sizeof(double*)*N*M);
    // double **sigma_invT = (double**)malloc(sizeof(double*)*N);
    // for (int i=0; i<N; i++)
    //     sigma_invT[i] = (double*)malloc(sizeof(double)*M);

	for(int i=0; i<N; i++){
		for(int j=0; j<M; j++) sigma_invT[i*M + j]=0;
	}

	e = eigenv_index.size()-1;

	for(int i=0; i<N;i++){
		if(eigenv_index[e].first<1e-5){
			sigma_invT[i*M + i]= 0;
		}
		else{
			sigma_invT[i*M + i]= 1/sqrt(eigenv_index[e].first);
		}
		e--;	
	}

	// double **temp = mat_mul(d, M, N, u, N, N);
	// double **v = mat_mul(temp, M, N, sigma_invT, N, M);
	// double **v_t = mat_transpose(v, M, M);
///////////////////////////////////////////////////////////////
    double *temp;
    temp = (double*)malloc(sizeof(double)*M*N);
    
    double *gpu_a1, *gpu_b1, *gpu_c1;
    hipMalloc((void **) &gpu_a1, sizeof(double)*M*N);
    hipMalloc((void **) &gpu_b1, sizeof(double)*N*N);
    hipMalloc((void **) &gpu_c1, sizeof(double)*M*N);

    // copying matrix d_t and d from host to device memory
    hipMemcpy(gpu_a1, d, sizeof(double)*M*N, hipMemcpyHostToDevice);
    hipMemcpy(gpu_b1, u, sizeof(double)*N*N, hipMemcpyHostToDevice);
    
    unsigned int g_rows1 = (M + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int g_cols1 = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid1(g_cols1, g_rows1);
    dim3 dimBlock1(BLOCK_SIZE, BLOCK_SIZE);
    
    gpu_matmul<<<dimGrid1, dimBlock1>>>(gpu_a1, gpu_b1, gpu_c1, M, N, N);
    hipMemcpy(temp, gpu_c1, sizeof(double)*M*N, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    // for(int i=0;i<N;i++){
    //   for(int j=0;j<N;j++){
    //       printf("%f ", (product1)[i*N+j]);
    //   }
    //   printf("\n");
    // }
    
///////////////////////////////////////////////////////////////

///////////////////////////////////////////////////////////////
    double *v;
    v = (double*)malloc(sizeof(double)*M*M);
    
    double *gpu_a2, *gpu_b2, *gpu_c2;
    hipMalloc((void **) &gpu_a2, sizeof(double)*M*N);
    hipMalloc((void **) &gpu_b2, sizeof(double)*N*M);
    hipMalloc((void **) &gpu_c2, sizeof(double)*M*M);

    // copying matrix d_t and d from host to device memory
    hipMemcpy(gpu_a2, temp, sizeof(double)*M*N, hipMemcpyHostToDevice);
    hipMemcpy(gpu_b2, sigma_invT, sizeof(double)*N*M, hipMemcpyHostToDevice);
    
    unsigned int g_rows2 = (M + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int g_cols2 = (M + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid2(g_cols2, g_rows2);
    dim3 dimBlock2(BLOCK_SIZE, BLOCK_SIZE);
    
    gpu_matmul<<<dimGrid2, dimBlock2>>>(gpu_a2, gpu_b2, gpu_c2, M, N, M);
    hipMemcpy(v, gpu_c2, sizeof(double)*M*M, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    // for(int i=0;i<N;i++){
    //   for(int j=0;j<N;j++){
    //       printf("%f ", (product1)[i*N+j]);
    //   }
    //   printf("\n");
    // }
    
///////////////////////////////////////////////////////////////




	// for(int i=0; i<M; i++){
	// 	for(int j=0; j<M; j++) printf("%f ", v_t[i][j]);
	// 	printf("\n");
	// }


	// for(int i=0; i<M; i++){
	// 	for(int j=0; j<M; j++) (*V_T)[i*M+j] = v_t[i][j];
	// }


    for(int i=0; i<M; i++){
        for(int j=0; j<M; j++) (*V_T)[i*M+j] = v[j*M + i];
    }
	// for(int i=0; i<M; i++){
	// 	for(int j=0; j<M; j++) printf("%f ", V_T[i][j]);
	// 	printf("\n");
	// }


	double num=0;
	int k=0;
	double sigmasqsum=0;
	for(k=0; k<N; k++){
		sigmasqsum += (*SIGMA)[k]*(*SIGMA)[k];
	}

	for(k=0; k<N; k++){
		num += ((*SIGMA)[k]*(*SIGMA)[k])/sigmasqsum;
		if(num >= retention/100.0){
			break;
		}
	}
    
    *K = k+1;

    // double **newU;
	// double **newU = (double**)malloc(sizeof(double*)*N*(k+1));
    double *newU = (double*)malloc(sizeof(double)*N*(k+1));
    // double **newU = (double**)malloc(sizeof(double*)*N);
    // for (int i=0; i<N; i++)
    //     newU[i] = (double*)malloc(sizeof(double)*(k+1));


    for(int i=0; i<N; i++){
    	for(int j=0;j<k+1;j++){
    		newU[i*(k+1) + j] = (u)[i*N + j];
    	}
    }


	// for(int i=0; i<N; i++){
	// 	for(int j=0; j<(k+1); j++) printf("%f ", newU[i][j]);
	// 	printf("\n");
	// }

    // double **d_hat = (double**)malloc(sizeof(double*)*M);
    // for (int i=0; i<(k+1); i++)
    //     d_hat[i] = (double*)malloc(sizeof(double)*(k+1));



    // d_hat = mat_mul(d, M, N, newU, N, (k+1));
///////////////////////////////////////////////////////////////
    double *d_hat = (double*)malloc(sizeof(double)*M*(k+1));
    // v = (double*)malloc(sizeof(double)*M*M);
    
    double *gpu_a3, *gpu_b3, *gpu_c3;
    hipMalloc((void **) &gpu_a3, sizeof(double)*M*N);
    hipMalloc((void **) &gpu_b3, sizeof(double)*N*(k+1));
    hipMalloc((void **) &gpu_c3, sizeof(double)*M*(k+1));

    // copying matrix d_t and d from host to device memory
    hipMemcpy(gpu_a3, d, sizeof(double)*M*N, hipMemcpyHostToDevice);
    hipMemcpy(gpu_b3, newU, sizeof(double)*N*(k+1), hipMemcpyHostToDevice);
    
    unsigned int g_rows3 = (M + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int g_cols3 = ((k+1) + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid3(g_cols3, g_rows3);
    dim3 dimBlock3(BLOCK_SIZE, BLOCK_SIZE);
    
    gpu_matmul<<<dimGrid3, dimBlock3>>>(gpu_a3, gpu_b3, gpu_c3, M, N, (k+1));
    hipMemcpy(d_hat, gpu_c3, sizeof(double)*M*(k+1), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    // for(int i=0;i<N;i++){
    //   for(int j=0;j<N;j++){
    //       printf("%f ", (product1)[i*N+j]);
    //   }
    //   printf("\n");
    // }
    
///////////////////////////////////////////////////////////////

	*D_HAT = (double*) malloc(sizeof(double) * M*(k+1));


	for(int i=0; i<M; i++){
    	for(int j=0;j<k+1;j++){
    		(*D_HAT)[i*(k+1)+j] = d_hat[i*(k+1) + j];
    	}
    }


}

